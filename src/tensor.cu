
#include <hip/hip_runtime.h>

float elapsedTime;

#define tpb 256
//FIXME: This was written to time the kernel, may need some work
#define LAUNCH_KERNEL(kernel, blocksPerGrid, threadsPerBlock, d_a, d_b, d_c, n)      \
    hipEvent_t start, stop;                                                         \
    hipEventCreate(&start);                                                         \
    hipEventCreate(&stop);                                                          \
    hipEventRecord(start, 0);                                                       \
    elementWiseMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n); \
    hipEventRecord(stop, 0);                                                        \
    hipEventSynchronize(stop);                                                      \
    hipEventElapsedTime(&elapsedTime, start, stop);                                 \
    hipEventDestroy(start);                                                         \
    hipEventDestroy(stop);

__global__ void elementWiseMultiplyKernel(const float *d_a, const float *d_b, float *d_c, int n)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n)
    {
        d_c[idx] = d_a[idx] * d_b[idx];
    }
}

hipError_t elementWiseMultiplyWrapper(const float *d_a, const float *d_b, float *d_c, int n)
{

    const int threadsPerBlock = tpb;
    const int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    // elementWiseMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);
    LAUNCH_KERNEL(elementWiseMultiplyKernel, blocksPerGrid, threadsPerBlock, d_a, d_b, d_c, n);
    // Check for errors in kernel launch
    hipError_t error = hipGetLastError();

    return error;
}